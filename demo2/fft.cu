#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hipfft/hipfft.h>
#include<hipblas.h>
#include<iostream>

#define BATCH 1
__global__ void real2complex(hipfftDoubleReal *in, hipfftDoubleComplex *out) {
	long i = blockIdx.x*blockDim.x + threadIdx.x;
	out[i] = make_hipDoubleComplex(in[i], 0);
}

void cuFFTR2C(hipfftHandle plan, hipfftDoubleReal *indata, hipfftDoubleComplex *outdata, int Nslice, int My, int Mx) {

	int nElem = Nslice*My*Mx;
	dim3 block(128);
	real2complex << < ((nElem + block.x - 1) / block.x), (block.x) >> > (indata, outdata);





	if (hipfftExecZ2Z(plan, outdata, outdata, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}
}

void cuFFTC2C(hipfftHandle plan, hipfftDoubleComplex *outdata, int Nslice, int My, int Mx) {




	if (hipfftExecZ2Z(plan, outdata, outdata, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}

}

void cuIFFTC2C(hipfftHandle plan, hipfftDoubleComplex *outdata, int Nslice, int My, int Mx) {   // because C-> C we only need 1 variable: outdata. 


	hipblasHandle_t handle;
	hipblasStatus_t status;
	double alpha = double(1) / (Nslice*My*Mx);



	status = hipblasCreate(&handle);



	if (hipfftExecZ2Z(plan, outdata, outdata, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}



	if (hipblasZdscal(handle, Nslice*My*Mx, &alpha, outdata, 1) != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "Cuda eror: Failed to synchronize ---- cuIFFTC2C\n");
		return;
	}

	hipblasDestroy(handle);

}

void cuFFTR2C_(hipfftHandle plan, hipfftDoubleReal *indata, hipfftDoubleComplex *outdata_, hipfftDoubleComplex *outdata, int Nslice, int My, int Mx) {

	int nElem = Nslice*My*Mx;
	dim3 block(128);
	real2complex << < ((nElem + block.x - 1) / block.x), (block.x) >> > (indata, outdata_);





	if (hipfftExecZ2Z(plan, outdata_, outdata, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}
}

void cuFFTC2C_(hipfftHandle plan, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, int Nslice, int My, int Mx) {




	if (hipfftExecZ2Z(plan, indata, outdata, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}

}

void cuIFFTC2C_(hipfftHandle plan, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, int Nslice, int My, int Mx) {   // because C-> C we only need 1 variable: outdata. 


	hipblasHandle_t handle;
	hipblasStatus_t status;
	double alpha = double(1) / (Nslice*My*Mx);



	status = hipblasCreate(&handle);



	if (hipfftExecZ2Z(plan, indata, outdata, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}



	if (hipblasZdscal(handle, Nslice*My*Mx, &alpha, outdata, 1) != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "Cuda eror: Failed to synchronize ---- cuIFFTC2C\n");
		return;
	}

	hipblasDestroy(handle);

}

void cuFFTR2C(hipStream_t stream, hipfftHandle plan, hipfftDoubleReal *indata, hipDoubleComplex *outdata, int Nslice, int My, int Mx) {

	int nElem = Nslice*My*Mx;
	dim3 block(64);
	real2complex << < (nElem / block.x), (block.x), 0, stream >> > (indata, outdata);





	if (hipfftExecZ2Z(plan, outdata, outdata, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}



}
void cuFFTC2C(hipStream_t stream, hipfftHandle plan, hipfftDoubleComplex *outdata, int Nslice, int My, int Mx) {




	if (hipfftExecZ2Z(plan, outdata, outdata, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}

}

void cuIFFTC2C(hipStream_t stream, hipfftHandle plan, hipblasHandle_t handle, hipfftDoubleComplex *outdata, int Nslice, int My, int Mx) {   // because C-> C we only need 1 variable: outdata. 


																																	//hipblasHandle_t handle;
																																	//hipblasStatus_t status;
	double alpha = float(1) / (Nslice*My*Mx);



	//status = hipblasCreate(&handle);



	if (hipfftExecZ2Z(plan, outdata, outdata, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z forward failed");
		return;
	}
	//if (hipDeviceSynchronize() != hipSuccess) {
	//	fprintf(stderr, "Cuda eror: Failed to synchronize\n");
	//	return;
	//}



	hipblasZdscal(handle, Nslice*My*Mx, &alpha, outdata, 1);

}