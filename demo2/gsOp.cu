#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "fft.h"
#include <stdlib.h>
#include <iostream>



__global__ void subsref(hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz) {
	long idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<nnz)
		outdata[idx] = indata[(long)S[idx]];
}
__global__ void subsasgn(hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz) {
	//*data must be memset
	long idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<nnz)
		outdata[(long)S[idx]] = indata[idx];

}

__global__ void subsref_(hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz) {
	long idx = blockIdx.x*blockDim.x + threadIdx.x;
	//if (idx<nnz/3)
		outdata[idx] = indata[(long)S[idx]];
		outdata[idx*2] = indata[(long)S[idx*2]];
		outdata[idx*3] = indata[(long)S[idx*3]];
}

__global__ void subsasgn_(hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz) {
	//*data must be memset
	long idx = blockIdx.x*blockDim.x + threadIdx.x;
	//if (idx<nnz)
		outdata[idx] = indata[(long)S[idx]];
		outdata[idx*2] = indata[(long)S[idx*2]];
		outdata[idx*3] = indata[(long)S[idx*3]];

}

__global__ void piecewiseMatMul(hipfftDoubleReal *ref, hipfftDoubleComplex *mat) {
	long idx = blockIdx.x*blockDim.x + threadIdx.x;
	mat[idx].x = ref[idx] * mat[idx].x;
	mat[idx].y = ref[idx] * mat[idx].y;
}

__global__ void piecewiseMatMul_(hipfftDoubleReal *ref, hipfftDoubleComplex *mat) {
	long idx = blockIdx.x*blockDim.x + threadIdx.x;
	mat[idx].x = ref[idx] * mat[idx].x;
	mat[idx].y = ref[idx] * mat[idx].y;

	mat[idx + 114688].x = ref[idx + 114688] * mat[idx + 114688].x;
	mat[idx + 114688].y = ref[idx + 114688] * mat[idx + 114688].y;

	mat[idx + 114688 * 2].x = ref[idx + 114688 * 2] * mat[idx + 114688 * 2].x;
	mat[idx + 114688 * 2].y = ref[idx + 114688 * 2] * mat[idx + 114688 * 2].y;

	mat[idx + 114688 * 3].x = ref[idx + 114688 * 3] * mat[idx + 114688 * 3].x;
	mat[idx + 114688 * 3].y = ref[idx + 114688 * 3] * mat[idx + 114688 * 3].y;
}

void AopReal(int Nslice, int My, int Mx, hipfftHandle plan, hipfftDoubleReal *indata, hipfftDoubleComplex *outdata_, hipfftDoubleComplex *outdata, double *S, long nnz) {



	cuFFTR2C(plan, indata, outdata_, Nslice, My, Mx);

	if (nnz % 128 == 0) {
		subsref << <nnz / 128, 128 >> > (outdata_, outdata, S, nnz);
	}
	else {
		subsref << <nnz / 96, 96 >> > (outdata_, outdata, S, nnz);
	}
	//hipDeviceSynchronize(); 
	//hipFree(outdata_); 


}
void AopCompl(int Nslice, int My, int Mx, hipfftHandle plan, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz) {
	//AWARE:  INPUT OF AOPCOMPL IS CHANGED, SO DON'T USE INPUT AGAIN!!!! IF YOU WANT TO USE, COPY IT TO ANOTHER VARIABLE. 
	//hipfftComplex *outdata_; 
	//hipMalloc(&outdata_, Nslice*My*Mx * sizeof(hipfftComplex));

	cuFFTC2C(plan, indata, Nslice, My, Mx);
	if (nnz % 128 == 0) {
		subsref << <nnz / 128, 128 >> > (indata, outdata, S, nnz);
	}
	else {
		subsref << <nnz / 96, 96 >> > (indata, outdata, S, nnz);
	}
	//hipDeviceSynchronize();


	//hipFree(outdata_); 

}
void Ahop(int Nslice, int My, int Mx, hipfftHandle plan, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz) {

	hipMemset(outdata, 0, Nslice*My*Mx * sizeof(hipfftDoubleComplex));

	if (nnz % 128 == 0) {
		subsasgn << <nnz / 128, 128 >> > (indata, outdata, S, nnz);
	}
	else {
		subsasgn << <nnz / 96, 96 >> > (indata, outdata, S, nnz);
	}

	//hipDeviceSynchronize();

	cuIFFTC2C(plan, outdata, Nslice, My, Mx);



}
void Bhop_gs(int Nslice, int My, int Mx, hipfftHandle plan, hipfftDoubleReal *ref, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata_, hipfftDoubleComplex *outdata, double *S, long nnz) {



	//hipMemset(outdata_, 0, Nslice*My*Mx * sizeof(hipfftDoubleComplex));


	Ahop(Nslice, My, Mx, plan, indata, outdata_, S, nnz);

	piecewiseMatMul << <(Nslice*My*Mx / 256), 256  >> >(ref, outdata_);
	//hipDeviceSynchronize();
	AopCompl(Nslice, My, Mx, plan, outdata_, outdata, S, nnz);
	//hipFree(outdata_);

}

void Iop_gs(int Nslice, int My, int Mx, hipfftHandle plan, hipfftDoubleReal *ref, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz) {

	//hipMemset(outdata, 0, Nslice*My*Mx * sizeof(hipfftDoubleComplex));

	Ahop(Nslice, My, Mx, plan, indata, outdata, S, nnz);
	piecewiseMatMul << <(Nslice*My*Mx / 256), 256 >> >(ref, outdata);

}

void AopReal_(int Nslice, int My, int Mx, hipfftHandle plan, hipfftDoubleReal *indata, hipfftDoubleComplex *outdata_, hipfftDoubleComplex *outdata_1, hipfftDoubleComplex *outdata, double *S, long nnz) {



	cuFFTR2C_(plan, indata, outdata_1, outdata_, Nslice, My, Mx);


	subsref << <3453, 64 >> > (outdata_, outdata, S, nnz);
	//hipDeviceSynchronize(); 
	//hipFree(outdata_); 


}
void AopCompl_(int Nslice, int My, int Mx, hipfftHandle plan, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, hipfftDoubleComplex *outdata_1, double *S, long nnz) {
	//AWARE:  INPUT OF AOPCOMPL IS CHANGED, SO DON'T USE INPUT AGAIN!!!! IF YOU WANT TO USE, COPY IT TO ANOTHER VARIABLE. 
	//hipfftComplex *outdata_; 
	//hipMalloc(&outdata_, Nslice*My*Mx * sizeof(hipfftComplex));

	cuFFTC2C_(plan, indata, outdata_1, Nslice, My, Mx);

	subsref << <3453, 64 >> > (outdata_1, outdata, S, nnz);
	//hipDeviceSynchronize();


	//hipFree(outdata_); 

}
void Ahop_(int Nslice, int My, int Mx, hipfftHandle plan, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, hipfftDoubleComplex *outdata_1, double *S, long nnz) {


	subsasgn << <3453, 64 >> > (indata, outdata_1, S, nnz);


	//hipDeviceSynchronize();

	cuIFFTC2C_(plan, outdata_1, outdata, Nslice, My, Mx);



}
void Bhop_gs_(int Nslice, int My, int Mx, hipfftHandle plan, hipfftDoubleReal *ref, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata_, hipfftDoubleComplex *outdata_1, hipfftDoubleComplex *outdata, double *S, long nnz) {



	hipMemset(outdata_, 0, Nslice*My*Mx * sizeof(hipfftDoubleComplex));


	Ahop_(Nslice, My, Mx, plan, indata, outdata_, outdata_1, S, nnz);

	piecewiseMatMul_ << <(Nslice*My*Mx / 256), 64 >> >(ref, outdata_);
	//hipDeviceSynchronize();
	AopCompl_(Nslice, My, Mx, plan, outdata_, outdata, outdata_1, S, nnz);
	//hipFree(outdata_);

}

void Iop_gs_(int Nslice, int My, int Mx, hipfftHandle plan, hipfftDoubleReal *ref, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, hipfftDoubleComplex *outdata_1, double *S, long nnz) {

	hipMemset(outdata, 0, Nslice*My*Mx * sizeof(hipfftDoubleComplex));

	Ahop_(Nslice, My, Mx, plan, indata, outdata, outdata_1, S, nnz);
	piecewiseMatMul_ << <(Nslice*My*Mx / 256), 64 >> >(ref, outdata);

}


void AopReal(int Nslice, int My, int Mx, hipfftHandle plan, hipblasHandle_t handle, hipStream_t stream, hipfftDoubleReal *indata, hipfftDoubleComplex *outdata_, hipfftDoubleComplex *outdata, double *S, long nnz, long numBlock, long blockSize) {



	cuFFTR2C(stream, plan, indata, outdata_, Nslice, My, Mx);


	subsref << <numBlock, blockSize, 0, stream >> > (outdata_, outdata, S, nnz);
	//hipDeviceSynchronize(); 
	//hipFree(outdata_); 


}

void AopCompl(int Nslice, int My, int Mx, hipfftHandle plan, hipblasHandle_t handle, hipStream_t stream, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz, long numBlock, long blockSize) {

	//hipfftComplex *outdata_; 
	//hipMalloc(&outdata_, Nslice*My*Mx * sizeof(hipfftComplex));

	cuFFTC2C(stream, plan, indata, Nslice, My, Mx);

	subsref << <numBlock, blockSize, 0, stream >> > (indata, outdata, S, nnz);
	//hipDeviceSynchronize();


	//hipFree(outdata_); 

}

void Ahop(int Nslice, int My, int Mx, hipfftHandle plan, hipblasHandle_t handle, hipStream_t stream, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz, long numBlock, long blockSize) {


	subsasgn << <numBlock, blockSize, 0, stream >> > (indata, outdata, S, nnz);


	//hipDeviceSynchronize();

	cuIFFTC2C(stream, plan, handle, outdata, Nslice, My, Mx);



}

void Bhop_gs(int Nslice, int My, int Mx, hipfftHandle plan, hipblasHandle_t handle, hipStream_t stream, hipfftDoubleReal *ref, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata_, hipfftDoubleComplex *outdata, double *S, long nnz, long numBlock, long blockSize) {



	hipMemset(outdata_, 0, Nslice*My*Mx * sizeof(hipfftComplex));


	Ahop(Nslice, My, Mx, plan, handle, stream, indata, outdata_, S, nnz, numBlock, blockSize);

	piecewiseMatMul_ << <(Nslice*My*Mx / 256), 64, 0, stream >> >(ref, outdata_);
	//hipDeviceSynchronize();
	AopCompl(Nslice, My, Mx, plan, handle, stream, outdata_, outdata, S, nnz, numBlock, blockSize);
	//hipFree(outdata_);

}
void Iop_gs(int Nslice, int My, int Mx, hipfftHandle plan, hipblasHandle_t handle, hipStream_t stream, hipfftDoubleReal *ref, hipfftDoubleComplex *indata, hipfftDoubleComplex *outdata, double *S, long nnz, long numBlock, long blockSize) {

	//hipMemset(outdata, 0, Nslice*My*Mx * sizeof(hipfftComplex));
	hipMemsetAsync(outdata, 0, Nslice*My*Mx * sizeof(hipfftComplex), stream);

	Ahop(Nslice, My, Mx, plan, handle, stream, indata, outdata, S, nnz, numBlock, blockSize);
	piecewiseMatMul_ << <(Nslice*My*Mx / 256), 64, 0, stream >> >(ref, outdata);

}